

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define DataType double

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len) {out[idx] = in1[idx] + in2[idx];}
  else {return;}
}

//@@ Insert code to implement timer start

double getTimer() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

//@@ Insert code to implement timer stop


int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  
  inputLength = atoi(argv[1]);
  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output

  int inputActualSize = inputLength * sizeof(DataType);
  hostInput1 = (DataType*) malloc(inputActualSize);
  if (hostInput1 == 0) {printf("hostInput1 malloc fail\n"); return 1;}
  hostInput2 = (DataType*) malloc(inputActualSize);
  if (hostInput2 == 0) {printf("hostInput2 malloc fail\n"); return 1;}
  hostOutput = (DataType*) malloc(inputActualSize);
  if (hostOutput == 0) {printf("hostOutput malloc fail\n"); return 1;}
  resultRef = (DataType*) malloc(inputActualSize);
  if (resultRef == 0) {printf("resultRef malloc fail\n"); return 1;}
  
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU

  for (int i = 0; i < inputLength; i++) {
      DataType randomNumber1 = (DataType) rand() / RAND_MAX;
      DataType randomNumber2 = (DataType) rand() / RAND_MAX;
      hostInput1[i] = randomNumber1;
      hostInput2[i] = randomNumber2;
      resultRef[i] = randomNumber1 + randomNumber2;
  }

  //@@ Insert code below to allocate GPU memory here


  hipMalloc(&deviceInput1, inputActualSize);
  hipMalloc(&deviceInput2, inputActualSize);
  hipMalloc(&deviceOutput, inputActualSize);

  //@@ Insert code to below to Copy memory to the GPU here

  double start = getTimer();
  hipMemcpy(deviceInput1, hostInput1, inputActualSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputActualSize, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  double duration = getTimer() - start;
  printf("Host to Device Time: %f\n", duration);

  //@@ Initialize the 1D grid and block dimensions here

  int threadPerBlock = 128;
  int blockNum = (inputLength + threadPerBlock - 1) / threadPerBlock;
  printf("threads per block: %i \n", threadPerBlock);
  printf("blocks num: %i \n", blockNum);
  
  //@@ Launch the GPU Kernel here

  start = getTimer();
  vecAdd <<<blockNum, threadPerBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
  hipDeviceSynchronize();
  duration = getTimer() - start;
  printf("CUDA Kernel: %f\n", duration);

  //@@ Copy the GPU memory back to the CPU here

  start = getTimer();
  hipMemcpy(hostOutput, deviceOutput, inputActualSize, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  duration = getTimer() - start;
  printf("Device to Host Time: %f\n", duration);

  //@@ Insert code below to compare the output with the reference

  bool allClose = true;
  for (int i = 0; i < inputLength; i++) {
    if (fabs(hostOutput[i] - resultRef[i]) > 1e-8) {
      allClose = false;
      break;
    }
  }

  allClose ? printf("All good!\n") : printf("Something not equal\n");

  //@@ Free the GPU memory here

  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  
  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);

  return 0;
}
